
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <sstream>
#include <limits>
#include <random>
#include <iomanip>

#define MAX_ATOMS 1032


// Must be <= 16kb
__constant__ float atoms[1032 * 4];


// Scatter approach requires atomic operations, gather approach is more parrallelizable

void __global__ c_energy(float* energy_grid, dim3 grid, float grid_spacing, float z, int num_atoms) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int atom_arr_dim = num_atoms * 4;
    int k = z / grid_spacing;

    float x = grid_spacing * (float) i;
    float y = grid_spacing * (float) j;
    
    float energy = 0.0f;

    for (int n = 0; n < atom_arr_dim; n += 4) {
        float dx = x - atoms[n];
        float dy = y - atoms[n + 1];
        float dz = z - atoms[n + 2];
        energy += atoms[n + 3] / sqrtf(dx * dx + dy * dy + dz * dz);
    }

    energy_grid[grid.x*grid.y*k + grid.x*j + i] += energy;
}

void __host__ generate_test_atoms() {
    std::ofstream file("atoms.txt"); // Output file
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis_xyz(0.0, 100);
    std::uniform_real_distribution<float> dis_potential(0.0, 10.0);

    if (file.is_open()) {
        for (int i = 0; i < 100; ++i) {
            float x = dis_xyz(gen);
            float y = dis_xyz(gen);
            float z = dis_xyz(gen);
            float potential = dis_potential(gen);

            file << std::fixed << std::setprecision(2)
                 << x << " " << y << " " << z << " " << potential << "\n";
        }
        file.close();
        std::cout << "Data file generated successfully.\n";
    } else {
        std::cerr << "Unable to create the file.\n";
    }
}

int __host__ main() {
    #ifdef __TEST__
    generate_test_atoms();
    #endif

    std::ifstream file("atoms.txt"); // Replace "input.txt" with your file name
    std::string line;
    float h_atoms[1032 * 4];
    int num_atoms = 0;
    // Assume all atoms are within this 128 x 128 x 128 cube
    int size_x = 128, size_y = 128, size_z = 128;

    while (std::getline(file, line)) {
        std::istringstream iss(line);
        float x, y, z, potential;
        if (iss >> x >> y >> z >> potential) {

            h_atoms[num_atoms * 4] = x;
            h_atoms[num_atoms * 4 + 1] = y;
            h_atoms[num_atoms * 4 + 2] = z;
            h_atoms[num_atoms * 4 + 3] = potential;
            num_atoms += 1;
        }
    }

    hipMemcpyToSymbol(HIP_SYMBOL(atoms), h_atoms, num_atoms * sizeof(float));

    float* h_energy_grid = new float[size_x * size_y * size_z], *d_energy_grid;
    dim3 grid(size_x, size_y, size_z);
    float grid_spacing = 0.5f;


    hipMalloc((void**)&d_energy_grid, size_x * size_y * size_z * sizeof(float));

    hipMemcpy(d_energy_grid, h_energy_grid, size_x * size_y * size_z * sizeof(float), hipMemcpyHostToDevice);

    for (int z = 0; z < size_z; z++) {
        c_energy<<<dim3(4, 4), dim3(32, 32)>>>(d_energy_grid, grid, grid_spacing, z, num_atoms);
    }

    hipFree(d_energy_grid);

    delete [] h_energy_grid;
}
